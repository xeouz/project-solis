#include "hip/hip_runtime.h"
#include "s_algo.cuh"
#include "s_image.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <memory.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#ifdef __INTELLISENSE__
    void __syncthreads();
    void __cudaPushCallConfiguration(dim3 gridDim, dim3 blockDim, size_t shared_mem=0, void* stream = 0);
#endif

__global__ void internal_blend_color_gpu(unsigned char* in_pixels, int width, double one_minus_alpha, double alpha, unsigned char r, unsigned char g, unsigned char b)
{
    int i = (blockIdx.y * blockDim.y + threadIdx.y);
    int j = (blockIdx.x * blockDim.x + threadIdx.x);

    int idx = (i*width+j)*BYTES_PER_PIXEL;
    in_pixels[idx  ] = ((alpha * r) + (one_minus_alpha * in_pixels[idx  ]));
    in_pixels[idx+1] = ((alpha * g) + (one_minus_alpha * in_pixels[idx+1]));
    in_pixels[idx+2] = ((alpha * b) + (one_minus_alpha * in_pixels[idx+2]));
}

namespace solis
{
    typedef struct
    {
        unsigned char pix[3];
    } Pixel;

    static unsigned char* in_pixels;
    static unsigned char* in_gpu_pixels;
    static unsigned int pixels_size;
    static dim3 grid;
    static dim3 threads_per_block;

    void wrapper_init_cuda(unsigned char* img, unsigned int height, unsigned int width)
    {
        pixels_size = height*width*BYTES_PER_PIXEL;
        
        // Copy image
        in_pixels = (unsigned char*)malloc(pixels_size);
        memcpy(in_pixels, img, pixels_size);

        // Init CUDA
        hipFree(0);

        hipMalloc((void**) &in_gpu_pixels, pixels_size);
        hipMemcpy(in_gpu_pixels, in_pixels, pixels_size, hipMemcpyHostToDevice);
        
        constexpr const int threads = 16;
        unsigned int num_blocks_w = width/threads + (bool)(width%threads);
        unsigned int num_blocks_h = height/threads + (bool)(height%threads);

        threads_per_block = dim3(threads, threads);
        grid = dim3(num_blocks_w, num_blocks_h);
    }
    void wrapper_blend_color_gpu(unsigned char* pixels, unsigned int height, unsigned int width, double alpha, unsigned char r, unsigned char g, unsigned char b)
    {
        double one_minus_alpha = (1 - alpha);

        internal_blend_color_gpu<<<grid, threads_per_block>>>(in_gpu_pixels, width, one_minus_alpha, alpha, r, g, b);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }
    unsigned char* wrapper_finish_cuda(unsigned int height, unsigned int width)
    {
        hipMemcpy(in_pixels, in_gpu_pixels, pixels_size, hipMemcpyDeviceToHost);
        hipFree(in_gpu_pixels);

        return in_pixels;
    }

}