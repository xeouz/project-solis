#include "hip/hip_runtime.h"
#include "s_algo.cuh"
#include "s_image.h"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <memory.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#ifdef __INTELLISENSE__
    void __syncthreads();
    void __cudaPushCallConfiguration(dim3 gridDim, dim3 blockDim, size_t shared_mem=0, void* stream = 0);
#endif

__global__ void internal_blend_color_gpu(unsigned char* in_pixels, int width, double one_minus_alpha, double alpha, unsigned char r, unsigned char g, unsigned char b)
{
    int i = (blockIdx.y * blockDim.y + threadIdx.y);
    int j = (blockIdx.x * blockDim.x + threadIdx.x);

    int idx = (i*width+j)*BYTES_PER_PIXEL;
    in_pixels[idx  ] = ((alpha * r) + (one_minus_alpha * in_pixels[idx  ]));
    in_pixels[idx+1] = ((alpha * g) + (one_minus_alpha * in_pixels[idx+1]));
    in_pixels[idx+2] = ((alpha * b) + (one_minus_alpha * in_pixels[idx+2]));
}

__global__ void internal_curand_init(hiprandState* current_state, int num_blocks_w, unsigned long seed)
{
    int i = (blockIdx.y * blockDim.y + threadIdx.y);
    int j = (blockIdx.x * blockDim.x + threadIdx.x);
    int idx = i*num_blocks_w+j;
    hiprand_init(seed, idx, 0, &current_state[idx]);
}

__global__ void internal_average_reduce_gpu(unsigned char* in_pixels, int charset_len, int font_size_y, int font_size_x, int width, int num_blocks_w, hiprandState* state)
{
    int i = (blockIdx.y * blockDim.y + threadIdx.y);
    int j = (blockIdx.x * blockDim.x + threadIdx.x);
    int rand_idx = i*num_blocks_w+j;
    int rand_num = ((int)(hiprand_uniform(&state[rand_idx]) * 1000)) % charset_len;
    i *= font_size_y;
    j *= font_size_x;

    int block_size_sq = font_size_y*font_size_x;
    int x, y, y_max=i+font_size_y, x_max=j+font_size_x, idx;
    int sum_r=0, sum_g=0, sum_b=0, avg_r, avg_g, avg_b;
    for(y=i; y<y_max; ++y)
    {
        for(x=j; x<x_max; ++x)
        {
            idx = (y*width+x)*BYTES_PER_PIXEL;
            sum_r += in_pixels[idx  ];
            sum_g += in_pixels[idx+1];
            sum_b += in_pixels[idx+2];

            in_pixels[idx  ] = 0;
            in_pixels[idx+1] = 0;
            in_pixels[idx+2] = 0;
        }
    }

    avg_r = sum_r/block_size_sq;
    avg_g = sum_g/block_size_sq;
    avg_b = sum_b/block_size_sq;

    idx = (i*width+j)*BYTES_PER_PIXEL;
    in_pixels[idx  ] = avg_r;
    in_pixels[idx+1] = avg_g;
    in_pixels[idx+2] = avg_b;
    in_pixels[idx+3] = rand_num;
}

namespace solis
{
    typedef struct
    {
        unsigned char pix[3];
    } Pixel;

    static unsigned char* in_pixels;
    static unsigned char* in_gpu_pixels;
    static unsigned int pixels_size;
    static unsigned int height;
    static unsigned int width;

    static unsigned int resized_height;
    static unsigned int resized_width;
    static unsigned int resized_size;

    static hiprandState* rand_state;

    unsigned int calculate_average_reduce_realloc_size(unsigned int old_size, unsigned int font_size_y, unsigned int font_size_x)
    {
        constexpr const int threads = DEFAULT_CUDA_NUM_THREADS;
        unsigned int advance_x = (threads*font_size_x), advance_y = (threads*font_size_y);
        unsigned int num_blocks_w = width/advance_x + (bool)(width%advance_x);
        unsigned int num_blocks_h = height/advance_y + (bool)(height%advance_y);

        resized_width = num_blocks_w * advance_x;
        resized_height = num_blocks_h * advance_y;

        resized_size = (resized_height*resized_width + (font_size_y*font_size_x)) * BYTES_PER_PIXEL;
        return resized_size;
    }

    void wrapper_init_cuda(unsigned char* img, unsigned int h, unsigned int w)
    {   
        resized_size = 0;
        resized_height = height = h; resized_width = width = w;
        resized_size = pixels_size = height*width*BYTES_PER_PIXEL;
        
        // Set image
        in_pixels = img;

        // Init CUDA
        hipFree(0);
        hipMalloc((void**) &in_gpu_pixels, pixels_size);
        hipMemcpy(in_gpu_pixels, in_pixels, pixels_size, hipMemcpyHostToDevice);
    }
    void wrapper_realloc_pixels_gpu(unsigned int new_size, unsigned int new_width)
    {
        unsigned char* buffer=(unsigned char*)malloc(new_size);
        memset(buffer, 0, new_size);
        unsigned int i, j;
        for(i=0; i<height; ++i)
        {
            for(j=0; j<width; ++j)
            {
                unsigned int idx1=(i*width+j)*BYTES_PER_PIXEL, idx2=(i*new_width+j)*BYTES_PER_PIXEL;
                buffer[idx2] = in_pixels[idx1];
                buffer[idx2+1] = in_pixels[idx1+1];
                buffer[idx2+2] = in_pixels[idx1+2];
            }
        }

        hipFree(in_gpu_pixels);
        hipMalloc((void**) &in_gpu_pixels, new_size);
        hipMemcpy(in_gpu_pixels, buffer, new_size, hipMemcpyHostToDevice);
        resized_size = new_size;
    }
    void wrapper_average_reduce_adjust_size(unsigned int old_size, unsigned int font_size_y,  unsigned int font_size_x)
    {
        unsigned int size=calculate_average_reduce_realloc_size(old_size, font_size_y, font_size_x);
        wrapper_realloc_pixels_gpu(size, resized_width);
    }
    void wrapper_blend_color_gpu(double alpha, unsigned char r, unsigned char g, unsigned char b)
    {
        constexpr const int threads = DEFAULT_CUDA_NUM_THREADS;
        unsigned int num_blocks_w = resized_width/threads + (bool)(resized_width%threads);
        unsigned int num_blocks_h = resized_height/threads + (bool)(resized_height%threads);
        dim3 threads_per_block = dim3(threads, threads);
        dim3 grid = dim3(num_blocks_w, num_blocks_h);
        double one_minus_alpha = (1 - alpha);

        internal_blend_color_gpu<<<grid, threads_per_block>>>(in_gpu_pixels, resized_width, one_minus_alpha, alpha, r, g, b);
    }
    void wrapper_average_reduce_gpu(const char* charset, unsigned int charset_len, unsigned int font_size_y, unsigned int font_size_x)
    {
        constexpr const int threads = DEFAULT_CUDA_NUM_THREADS;
        unsigned int advance_x = (threads*font_size_x), advance_y = (threads*font_size_y);
        unsigned int num_blocks_w = resized_width/advance_x + (bool)(resized_width%advance_x);
        unsigned int num_blocks_h = resized_height/advance_y + (bool)(resized_height%advance_y);
        dim3 threads_per_block = dim3(threads, threads);
        dim3 grid = dim3(num_blocks_w, num_blocks_h);

        unsigned int charset_size = sizeof(char) * charset_len;

        hipMalloc(&rand_state, num_blocks_h * num_blocks_w * threads*threads * sizeof(hiprandState));
        internal_curand_init<<<grid, threads_per_block>>>(rand_state, num_blocks_w, 1);

        char* charset_gpu;
        hipMalloc(&charset_gpu, charset_size);
        hipMemcpy(charset_gpu, charset, charset_size, hipMemcpyHostToDevice);
        internal_average_reduce_gpu<<<grid, threads_per_block>>>(in_gpu_pixels, charset_len, font_size_y, font_size_x, resized_width, num_blocks_w, rand_state);
        hipFree(&charset_gpu);
    }
    unsigned char* wrapper_finish_cuda()
    {
        if(false)
            hipMemcpy(in_pixels, in_gpu_pixels, pixels_size, hipMemcpyDeviceToHost);
        else
        {
            unsigned char* buffer=(unsigned char*)malloc(resized_size), *gpu_pix, *res_pix;
            unsigned int i, j;
            hipMemcpy(buffer, in_gpu_pixels, resized_size, hipMemcpyDeviceToHost);

            for(i=0; i<height; ++i)
            {
                for(j=0; j<width; ++j)
                {
                    gpu_pix = buffer+((i*resized_width+j)*BYTES_PER_PIXEL); res_pix = in_pixels+((i*width+j)*BYTES_PER_PIXEL);
                    res_pix[0] = gpu_pix[0];
                    res_pix[1] = gpu_pix[1];
                    res_pix[2] = gpu_pix[2];                    
                }
            }
            free(buffer);
        }
        hipFree(in_gpu_pixels);

        return in_pixels;
    }

}